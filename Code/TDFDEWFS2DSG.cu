
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <math.h>
#include <stdio.h>

#define PI 3.141592654
using namespace std;

const int nOrder = 3;
const int nTimePreSnap = 100;

typedef struct {
  int nx, nz;
  int Nx, Nz;
  int sx, sz;
  int npx, npz;
  float dx, dz;
} dim;
typedef struct {
  float *vp, *vs, *rho;
} media;
typedef struct {
  float *vxx, *vxz, *vzx, *vzz,
        *txxx, *txxz, *tzzx, *tzzz,
        *txzx, *txzz;
  float *vxt, *vzt, *txxt, *tzzt, *txzt;
} wave;
typedef struct {
  float dt;
  float d0x, d0z;
  float C[nOrder];
} coeff;
typedef struct {
  float *f0, *f1, *f2, *f3, *f4, *f5, *f6, *f7, *f8, *f9;
} factor;

__global__ void pre_eval(wave W, media M, dim D, coeff C, factor F) {
  int ix = threadIdx.x + blockIdx.x*blockDim.x;
  int iz = threadIdx.y + blockIdx.y*blockDim.y;
  int idx = iz*D.Nx + ix;
  float dpmlx = 0.0, dpmlz = 0.0;
  float lambda, mu;

  if(ix < D.Nx && iz < D.Nz) {
    mu = M.rho[idx]*M.vs[idx]*M.vs[idx];
    lambda = M.rho[idx]*M.vp[idx]*M.vp[idx] - 2*mu;
    if(ix < D.npx + nOrder && ix >= nOrder)
      dpmlx = C.d0x*pow(1.0*(D.npx + nOrder - ix)/D.npx, 2);
    if(ix >= D.Nx - D.npx - nOrder && ix < D.Nx - nOrder)
      dpmlx = C.d0x*pow(1.0*(ix + D.npx + nOrder + 1 - D.Nx)/D.npx, 2);
    if(iz < D.npz + nOrder && iz >= nOrder)
      dpmlz = C.d0z*pow(1.0*(D.npz + nOrder - iz)/D.npz, 2);
    if(iz >= D.Nz - D.npz - nOrder && iz < D.Nz - nOrder)
      dpmlz = C.d0z*pow(1.0*(iz + D.npz + nOrder + 1 - D.Nz)/D.npz, 2);

    F.f0[idx] = (2 - C.dt*dpmlx)/(2 + C.dt*dpmlx);
    F.f1[idx] = (2 - C.dt*dpmlz)/(2 + C.dt*dpmlz);
    F.f2[idx] = 2*C.dt/(2 + C.dt*dpmlx)/M.rho[idx]/D.dx;
    F.f3[idx] = 2*C.dt/(2 + C.dt*dpmlz)/M.rho[idx]/D.dz;
    F.f4[idx] = 2*C.dt/(2 + C.dt*dpmlx)*(lambda + 2*mu)/D.dx;
    F.f5[idx] = 2*C.dt/(2 + C.dt*dpmlz)*lambda/D.dz;
    F.f6[idx] = 2*C.dt/(2 + C.dt*dpmlx)*lambda/D.dx;
    F.f7[idx] = 2*C.dt/(2 + C.dt*dpmlz)*(lambda + 2*mu)/D.dz;
    F.f8[idx] = 2*C.dt/(2 + C.dt*dpmlx)*mu/D.dx;
    F.f9[idx] = 2*C.dt/(2 + C.dt*dpmlz)*mu/D.dz;

    W.vxx [idx] = 0.0; W.vxz [idx] = 0.0; W.vxt [idx] = 0.0;
    W.vzx [idx] = 0.0; W.vzz [idx] = 0.0; W.vzt [idx] = 0.0;
    W.txxx[idx] = 0.0; W.txxz[idx] = 0.0; W.txxt[idx] = 0.0;
    W.tzzx[idx] = 0.0; W.tzzz[idx] = 0.0; W.tzzt[idx] = 0.0;
    W.txzx[idx] = 0.0; W.txzz[idx] = 0.0; W.txzt[idx] = 0.0;
  }
}

__global__ void vel_eval(wave W, dim D, coeff C, factor F, int sidx) {
  int ix = threadIdx.x + blockIdx.x*blockDim.x;
  int iz = threadIdx.y + blockIdx.y*blockDim.y;
  int idx = iz*D.Nx + ix;
  int i;
  float Psum;
  
  if(ix >= nOrder && ix < D.Nx - nOrder && iz >= nOrder && iz < D.Nz - nOrder) {
    Psum = 0.0;
    for(i = 0; i < nOrder; i++)
      Psum += C.C[i]*(W.txxt[idx + i] - W.txxt[idx - i - 1]);
    W.vxx[idx] = F.f0[idx]*W.vxx[idx] + F.f2[idx]*Psum;
    Psum = 0.0;
    for(i = 0; i < nOrder; i++)
      Psum += C.C[i]*(W.txzt[idx + i*D.Nx] - W.txzt[idx - (i + 1)*D.Nx]);
    W.vxz[idx] = F.f1[idx]*W.vxz[idx] + F.f3[idx]*Psum;
    W.vxt[idx] = W.vxx[idx] + W.vxz[idx];

    Psum = 0.0;
    for(i = 0; i < nOrder; i++)
      Psum += C.C[i]*(W.txzt[idx + i + 1] - W.txzt[idx - i]);
    W.vzx[idx] = F.f0[idx]*W.vzx[idx] + F.f2[idx]*Psum;
    Psum = 0.0;
    for(i = 0; i < nOrder; i++)
      Psum += C.C[i]*(W.tzzt[idx + (i + 1)*D.Nx] - W.tzzt[idx - i*D.Nx]);
    W.vzz[idx] = F.f1[idx]*W.vzz[idx] + F.f3[idx]*Psum;
    W.vzt[idx] = W.vzx[idx] + W.vzz[idx];
  }
}

__global__ void str_eval(wave W, dim D, coeff C, int sidx, float srclet, factor F) {
  int ix = threadIdx.x + blockIdx.x*blockDim.x;
  int iz = threadIdx.y + blockIdx.y*blockDim.y;
  int idx = iz*D.Nx + ix;
  int i;
  float Psum;

  if(ix >= nOrder && ix < D.Nx - nOrder && iz >= nOrder && iz < D.Nz - nOrder) {
    Psum = 0.0;
    for(i = 0; i < nOrder; i++)
      Psum += C.C[i]*(W.vxt[idx + i + 1] - W.vxt[idx - i]);
    W.txxx[idx] = F.f0[idx]*W.txxx[idx] + F.f4[idx]*Psum;
    W.tzzx[idx] = F.f0[idx]*W.tzzx[idx] + F.f6[idx]*Psum;
    Psum = 0.0;
    for(i = 0; i < nOrder; i++)
      Psum += C.C[i]*(W.vzt[idx + i*D.Nx] - W.vzt[idx - (i + 1)*D.Nx]);
    W.txxz[idx] = F.f1[idx]*W.txxz[idx] + F.f5[idx]*Psum;
    W.tzzz[idx] = F.f1[idx]*W.tzzz[idx] + F.f7[idx]*Psum;

    Psum = 0.0;
    for(i = 0; i < nOrder; i++)
      Psum += C.C[i]*(W.vzt[idx + i] - W.vzt[idx - i - 1]);
    W.txzx[idx] = F.f0[idx]*W.txzx[idx] + F.f8[idx]*Psum;
    Psum = 0.0;
    for(i = 0; i < nOrder; i++)
      Psum += C.C[i]*(W.vxt[idx + (i + 1)*D.Nx] - W.vxt[idx - i*D.Nx]);
    W.txzz[idx] = F.f1[idx]*W.txzz[idx] + F.f9[idx]*Psum;
    W.txzt[idx] = W.txzx[idx] + W.txzz[idx];

    if(idx == sidx) {
      W.txxx[idx] += srclet/4;
      W.txxz[idx] += srclet/4;
      W.tzzx[idx] += srclet/4;
      W.tzzz[idx] += srclet/4;
    }
    W.txxt[idx] = W.txxx[idx] + W.txxz[idx];
    W.tzzt[idx] = W.tzzx[idx] + W.tzzz[idx];
  }
}

void exp_wave(dim D, char *filename, float *P) {
//
  FILE *fp = fopen(filename, "wb");
    fwrite(&D.nx, sizeof(float), 1, fp);
    fwrite(&D.nz, sizeof(float), 1, fp);
    for(int i = 0; i < D.nz; i++) {
      fwrite(&P[(i + D.npz + nOrder)*D.Nx + D.npx + nOrder], sizeof(float), D.nx, fp);
    }
  fclose(fp);
/*
  FILE *fp = fopen(filename, "wt");
    for(int i = 0; i < D.nz; i++) {
      for(int j = 0; j < D.nx; j++)
        fprintf(fp, "%lf, ", (double)P[(i + D.npz + nOrder)*D.Nx + D.npx + nOrder + j]);
      fprintf(fp, "\n");
    }
  fclose(fp);
*/
}

int main(int argc, char *argv[]) {

  int nx = 500, nz = 600;
  int npmlx = 20, npmlz = 20;
  int sx = 80, sz = 80;
  float dx = 5.0, dz = 5.0;
  int nt = 1000;
  float dt = 1.0e-3;
  int nppw = 12;
  float ampl = 1.0e0;

  wave W; media M; dim D; coeff C; factor F;

  int Nx, Nz;
  size_t memSize;
  int i, j;

  cout << "Input nt = ";
  cin >> nt;
  
  int prod1, prod2;
  for(int m = 1; m < nOrder + 1; m++) {
    prod1 = 1;
    for(i = 1; i <= nOrder; i++)
      if(i != m) prod1 *= (2*i - 1)*(2*i - 1);
    prod2 = 1;
    for(i = 1; i <= nOrder; i++)
      if(i != m) prod2 *= abs((2*m - 1)*(2*m - 1) - (2*i - 1)*(2*i - 1));
    C.C[m - 1] = pow(-1.0, m + 1)*prod1/(2*m - 1)/prod2;
  }

  Nx = nx + 2*npmlx + 2*nOrder;
  Nz = nz + 2*npmlz + 2*nOrder;
  memSize = Nx*Nz*sizeof(float);

  D.nx = nx; D.nz = nz;
  D.Nx = Nx; D.Nz = Nz;
  D.sx = sx; D.sz = sz;
  D.npx = npmlx; D.npz = npmlz;
  D.dx = dx; D.dz= dz;

  float *Vp  = (float*) malloc(memSize);
  float *Vs  = (float*) malloc(memSize);
  float *Rho = (float*) malloc(memSize);
  for(i = 0; i < Nx*Nz; i++) {
    Vp [i] = 2000.0;
    Vs [i] = 1000.0;
    Rho[i] = 1000.0;
  }

  hipMalloc((float**) &M.vp , memSize);
  hipMalloc((float**) &M.vs , memSize);
  hipMalloc((float**) &M.rho, memSize);
  hipMemcpy(M.vp , Vp , memSize, hipMemcpyHostToDevice);
  hipMemcpy(M.vs , Vs , memSize, hipMemcpyHostToDevice);
  hipMemcpy(M.rho, Rho, memSize, hipMemcpyHostToDevice);

  float *vsmin = Vs, *vsmax = Vs;
  for(i = 1; i < Nx*Nz; i++) {
    if(*vsmin > Vs[i]) vsmin = &Vs[i];
    if(*vsmax < Vs[i]) vsmax = &Vs[i];
  }

  float f0, t0;
  f0 = (*vsmin)/(max(dx, dz)*nppw);
  t0 = 1.0/f0;
  C.dt = dt;
  C.d0x = 3*(*vsmax)/dx*(8.0/15 - 3.0/100*npmlx + 1.0/1500*npmlx*npmlx);
  C.d0z = 3*(*vsmax)/dz*(8.0/15 - 3.0/100*npmlz + 1.0/1500*npmlz*npmlz);

  hipMalloc((float**) &F.f0, memSize);
  hipMalloc((float**) &F.f1, memSize);
  hipMalloc((float**) &F.f2, memSize);
  hipMalloc((float**) &F.f3, memSize);
  hipMalloc((float**) &F.f4, memSize);
  hipMalloc((float**) &F.f5, memSize);
  hipMalloc((float**) &F.f6, memSize);
  hipMalloc((float**) &F.f7, memSize);
  hipMalloc((float**) &F.f8, memSize);
  hipMalloc((float**) &F.f9, memSize);

  hipMalloc((float**) &W.vxx , memSize);
  hipMalloc((float**) &W.vxz , memSize);
  hipMalloc((float**) &W.vzx , memSize);
  hipMalloc((float**) &W.vzz , memSize);
  hipMalloc((float**) &W.txxx, memSize);
  hipMalloc((float**) &W.txxz, memSize);
  hipMalloc((float**) &W.tzzx, memSize);
  hipMalloc((float**) &W.tzzz, memSize);
  hipMalloc((float**) &W.txzx, memSize);
  hipMalloc((float**) &W.txzz, memSize);
  hipMalloc((float**) &W.vxt , memSize);
  hipMalloc((float**) &W.vzt , memSize);
  hipMalloc((float**) &W.txxt, memSize);
  hipMalloc((float**) &W.tzzt, memSize);
  hipMalloc((float**) &W.txzt, memSize);

  dim3 Block(32, 16);
  dim3 Grid(ceil(1.0*Nx/Block.x), ceil(1.0*Nz/Block.y));
  
  cout << "Block = " << Block.x << " " << Block.y << endl;
  cout << "Grid = " << Grid.x << " " << Grid.y << endl;

  float *P;
  P = (float*) malloc(memSize);

  float srclet;
  int sidx = npmlx + sx + nOrder - 1 + (npmlz + sz + nOrder - 1)*D.Nx;
  pre_eval <<< Grid, Block >>> (W, M, D, C, F);

  int it = 0;
  char file[200];
  for(i = 0; i*nTimePreSnap < nt; i++) {
    printf("calculating and exporting for step %5d ...\n", it);
    for(j = 0; j < nTimePreSnap; j++, it++) {
      if(it > nt) break;
      srclet = ampl*(1 - 2*pow(PI*f0*(dt*it - t0), 2))*exp( - pow(PI*f0*(dt*it - t0), 2));

      str_eval <<< Grid, Block >>> (W, D, C, sidx, srclet, F);
      vel_eval <<< Grid, Block >>> (W, D, C, F, sidx);

      if((it - 1)%nTimePreSnap == 0) {
        hipDeviceSynchronize();
        sprintf(file, "./data/P%05d.bin", it - 1);
        hipMemcpy(P, W.txxt, memSize, hipMemcpyDeviceToHost);
        exp_wave(D, file, P);
      }
    }
  }

  free(Vp); free(Vs); free(Rho);
  hipFree(M.vp); hipFree(M.vs); hipFree(M.rho);

  free(P);

  hipFree(W.vxx ); hipFree(W.vxz ); hipFree(W.vxt );
  hipFree(W.vzx ); hipFree(W.vzz ); hipFree(W.vzt );
  hipFree(W.txxx); hipFree(W.txxz); hipFree(W.txxt);
  hipFree(W.tzzx); hipFree(W.tzzz); hipFree(W.tzzt);
  hipFree(W.txzx); hipFree(W.txzz); hipFree(W.txzt);

  return 0;
}
